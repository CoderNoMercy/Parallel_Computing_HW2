#include "hip/hip_runtime.h"
#include "utils.cuh"

__global__ void matvec_kernel(float (*L)[114], float* tau_a, float* matvec_out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i=idx+1; i<n; i+= blockDim.x * gridDim.x){
        matvec_out[i] = 0
        for (int j=0; j<=114; j++){
            matvec_out[i] += L[i][j] * tau_a[j];
        }
    }
}

void matvec_func(float (*L)[114], const boost::array<float, 114>& tau_h, const boost::array<float, 114>& out_h) {
    
    const int threads_per_block = 8;
    int numBlocks = (114 + threads_per_block  - 1) / threads_per_block;
    
    size_t buf_L = 114*114*sizeof(float);
    size_t buf_tau = 114*sizeof(float);
   
    float (*L_d)[114];
    if (hipMalloc((void**)&(*L_d)[114], buf_L) != hipSuccess) {
        perror("cuda malloc failed!");
    }
    float* tau_d;
    if (hipMalloc((void**)&tau_d, buf_tau) != hipSuccess) {
        perror("cuda malloc failed!");
    }
    float* out_d;
    if (hipMalloc((void**)&out_d, buf_tau) != hipSuccess) {
        perror("cuda malloc failed!");
    }
   
    hipMemcpy(L_d, L_h, buf_L, hipMemcpyHostToDevice);
    hipMemcpy(tau_d, tau_h.data(), buf_tau, hipMemcpyHostToDevice);
    hipMemcpy(out_d, out_h.data(), buf_tau, hipMemcpyHostToDevice);

    hipEvent_t start1, end1;
    hipEventCreate(&start1);
    hipEventCreate(&end1);
    hipEventRecord(start1, 0);

    matvec_kernel<<<numBlocks, threads_per_block>>>(L_d, tau_d, out_d);

    hipEventRecord(end1, 0);
    hipEventSynchronize(end1);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start1, end1);

    hipMemcpy(out_h.data(), out_d, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(L_d);
    hipFree(tau_d);
    
    printf("cuda run time %3.1f ms\n", rank, milliseconds);
}
