#include "hip/hip_runtime.h"
#include "parscan.cuh"

__global__ void sequential_sum(float* a, float* s, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i=idx+1; i<n; i+= blockDim.x * gridDim.x){
        for (int j=0; j<=i; j++){
            s[i] += a[j];
        }
    }
}

__global__ void pair_sum(float* a, float* b, int m){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i=idx; i<m; i+=blockDim.x * gridDim.x){
        b[i] = a[i*2] + a[i*2+1];
    }
}

__global__ void assign_ans(float* a, float* c, float* s, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i=idx; i<n; i+=blockDim.x * gridDim.x){
        if (i%2==1) s[i] = c[(i-1)/2];
	else s[i] = a[i] + c[i/2 - 1];
    }
}

__global__ void sequential_add(float* a, int n, float addition) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i=idx; i<n; i+= blockDim.x * gridDim.x){
        a[i] += addition;
    }
}

void parscan_mpicuda(int n, int rank) {
    
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    const int threads_per_block = 8;
    int numBlocks = (n + threads_per_block  - 1) / threads_per_block;
    
    int m = n / 2;
    size_t buf_size = n*sizeof(float);
    size_t buf_m = m*sizeof(float);
    float* a = (float*)malloc(buf_size);
    float* b = (float*)malloc(buf_m);
    float* s = (float*)malloc(buf_size);

    MPI_Status mpi_status;
    float* da;
    hipMalloc((void**)&da, buf_size);
    float* db;
    hipMalloc((void**)&db, buf_m);
    MPI_Recv(a, n, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &mpi_status);
    hipMemcpy(da, a, buf_size, hipMemcpyHostToDevice);
    pair_sum<<<numBlocks, threads_per_block>>>(da, db, m);
    hipMemcpy(b, db, buf_m, hipMemcpyDeviceToHost);
    
    //float* c = parscan_cuda(b, m);

    float* dc;
    hipMalloc((void**)&dc, buf_m);
    sequential_sum<<<numBlocks, threads_per_block>>>(db, dc, n);
    //hipMemcpy(dc, c, buf_m, hipMemcpyHostToDevice);

    float* ds;
    hipMalloc((void**)&ds, buf_size);
    assign_ans<<<numBlocks, threads_per_block>>>(da, dc, ds, n);

    float last_element = 0;
    hipMemcpy(&last_element, ds + n - 1, sizeof(float), hipMemcpyDeviceToHost);

    MPI_Send(&last_element, 1, MPI_FLOAT, 0, 102, MPI_COMM_WORLD);
    if (rank != 1) {
        float addition=0;
        MPI_Recv(&addition, 1, MPI_FLOAT, 0, 103, MPI_COMM_WORLD, &mpi_status);
        sequential_add<<<numBlocks, threads_per_block>>>(ds, n, addition);
    }
    
    hipMemcpy(s, ds, buf_size, hipMemcpyDeviceToHost);
    s[0] = a[0];
    MPI_Send(s, n, MPI_FLOAT, 0, 101, MPI_COMM_WORLD);

    hipFree(da);
    hipFree(ds);
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    float milliseconds = 0;
    cout << hipEventElapsedTime(&milliseconds, start, end);
}

float* parscan_cuda(float* a, int n){
    if (n==1) return a;
    int threads_per_block=8;
    int numBlocks = (n + threads_per_block - 1) / threads_per_block;
    
    int m = n/2;
    float* b = (float*)malloc(m*sizeof(float));
    float* da;
    hipMalloc((void**)&da, n*sizeof(float));
    float* db;
    hipMalloc((void**)&db, m*sizeof(float));
    hipMemcpy(da, a, n*sizeof(float), hipMemcpyHostToDevice);
    pair_sum<<<numBlocks, threads_per_block>>>(da, db, m);
    hipMemcpy(b, db, m*sizeof(float), hipMemcpyDeviceToHost);
    
    float* c = parscan_cuda(b, m);
    float* dc;
    hipMalloc((void**)&dc, m * sizeof(float));
    hipMemcpy(dc, c, m*sizeof(float), hipMemcpyHostToDevice);
    float* ds;
    hipMalloc((void**)&ds, n * sizeof(float));
    assign_ans<<<numBlocks, threads_per_block>>>(da, dc, ds, n);
    float* s = (float*)malloc(n * sizeof(float));
    hipMemcpy(s, ds, n*sizeof(float), hipMemcpyDeviceToHost);
    return s;
}


void seq_scan_mpicuda(int n, int rank) {
    
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    const int threads_per_block = 8;
    int numBlocks = (n + threads_per_block  - 1) / threads_per_block;
    
    size_t buf_size = n*sizeof(float);
    float* a = (float*)malloc(buf_size);
    float* s = (float*)malloc(buf_size);

    MPI_Status mpi_status;
    float* da;
    hipMalloc((void**)&da, buf_size);
    MPI_Recv(a, n, MPI_FLOAT, 0, 100, MPI_COMM_WORLD, &mpi_status);
    // for (int i=0; i < n; i++){
    //     printf("cuda program input:%f in rank %d", a[i], rank);
    // }
    hipMemcpy(da, a, buf_size, hipMemcpyHostToDevice);
    float* ds;
    hipMalloc((void**)&ds, buf_size);
    sequential_sum<<<numBlocks, threads_per_block>>>(da, ds, n);

    float last_element =0;
    hipMemcpy(&last_element, ds + n - 1, sizeof(float), hipMemcpyDeviceToHost);

    MPI_Send(&last_element, 1, MPI_FLOAT, 0, 102, MPI_COMM_WORLD);
    if (rank != 1) {
        float addition=0;
        MPI_Recv(&addition, 1, MPI_FLOAT, 0, 103, MPI_COMM_WORLD, &mpi_status);
        sequential_add<<<numBlocks, threads_per_block>>>(ds, n, addition);
    }
    
    hipMemcpy(s, ds, buf_size, hipMemcpyDeviceToHost);
    s[0] = a[0];
    // printf("before sending local output to master\n");
    MPI_Send(s, n, MPI_FLOAT, 0, 101, MPI_COMM_WORLD);
    // for (int i=0; i < n; i++){
    //     printf("cuda program output:%f in rank %d", s[i], rank);
    // }
    // printf("after sending local output to master\n");

    hipFree(da);
    hipFree(ds);
    
    hipEventRecord(end);
    hipEventSynchronize(end);
    float milliseconds = 0;
    cout << hipEventElapsedTime(&milliseconds, start, end);    
}

float* seq_scan_cuda(float *a, int n) {
    
    if (n == 1) {
        return a;
    }
    
    const int threads_per_block = 8;
    int numBlocks = (n + threads_per_block  - 1) / threads_per_block;
    
    size_t buf_size = n*sizeof(float);
    float* s = (float*)malloc(buf_size);
    s[0] = a[0];

    float* da;
    hipMalloc((void**)&da, buf_size);
    hipMemcpy(da, a, buf_size, hipMemcpyHostToDevice);
    float* ds;
    hipMalloc((void**)&ds, buf_size);
    hipMemcpy(ds, s, buf_size, hipMemcpyHostToDevice);
    sequential_sum<<<numBlocks, threads_per_block>>>(da, ds, n);
    hipMemcpy(s, ds, buf_size, hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(ds);
    hipDeviceReset();
    return s;
    
}
